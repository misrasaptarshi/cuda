

// Matrices are stored in row-major order
// M(row,col) - *(M.elements + row*M.width + col)

typedef struct{
	int width;
	int height;
	float *elements;
}Matrix;

//Thread block size
#define BLOCK_SIZE 16

//Declaration of the matrix multiplication kernel
__global__ void MatMulkernel(Matrix A,  Matrix B, Matrix C)
{
	//Each thread computes one element of C by accumulating results into Cvalue

	float Cvalue=0;
	int row = blockIdx.y*blockDim.y + threadIdx.y;
	int col = blockIdx.x*blockDim.x + threadIdx.x;
	for (int e=0; e<A.width; ++e)
		Cvalue+= A.elements[row*A.width+e] * B.elements[e*B.width+col];
 	C.elements[row*C.width+col] = Cvalue;
}

// Matrix multiplication - Host code
//Matrix dimensions are assumed to be multiples of BLOCK_SIZE


Matrix MatMul(const Matrix A, const Matrix B, Matrix C)
{

	//Load A and B to device memory
	Matrix d_A;
	d_A.width = A.width; d_A.height = A.height;
	size_t size = A.width * A.height * sizeof(float);
	hipMalloc(&d_A.elements,size);
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);

	Matrix d_B;
	d_B.width = B.width; d_B.height = B.height;
	size = B.width*B.height*sizeof(float);
	hipMalloc(&d_B.elements, size);
	hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);

	//Allocate c in device memory
	Matrix d_C;
	d_C.width = C.width; d_C.height = C.height;
	size = C.width*C.height*sizeof(float);
	hipMalloc(&d_C.elements, size);

	//Invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(B.width/dimBlock.x, A.height/dimBlock.y);
	MatMulkernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);

	// Read C from device memory
	hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);

	return C;
}



#include <hip/hip_runtime.h>
#include<stdio.h>
int main()
{
	/*Matrix a;
	Matrix b;
	Matrix c;
	a.width = 2; a.height = 2;
	float size = 4*sizeof(float*);
	
	b.width = 2; b.height = 2;
	c.width = 2; c.height = 2;
	a.elements = (float*)malloc(size);
        b.elements = (float*)malloc(size);
	c.elements = (float*)malloc(size);

	//a.elements[400];
	//b.elements[400];
	//c.elements[400];
	
	int i,j;
	for (i=0;i<2;i++){
		for (j=0;j<2;j++){
			*(a.elements + i*2 + j) = (i+j);
		}
	}

	for (i=0;i<2;i++){
		for (j=0;j<2;j++){
			*(b.elements + i*2 + j) = (i+j);
		}
	}

	MatMul(a,b,c);

	for (i=0;i<2;i++){
		for (j=0;j<2;j++){
			printf ("%f \n", *(c.elements + i*2 + j));
		}
	}*/

	Matrix a;
	Matrix b;
	Matrix c;

	//a = (Matrix*)malloc(sizeof(Matrix));
	//b = (Matrix*)malloc(sizeof(Matrix));
	//c = (Matrix*)malloc(sizeof(Matrix));

	a.width=2; a.height=2;
	b.width=2; b.height=2;
	c.width=2; c.height=2;

	a.elements = (float*)malloc(4*sizeof(float));
	b.elements = (float*)malloc(4*sizeof(float));
	c.elements = (float*)malloc(4*sizeof(float));

	//a.elements[0] = 0; a.elements[1] = 1; a.elements[2] = 1; a.elements[3] = 2;
	//b.elements[0] = 0; b.elements[1] = 1; b.elements[2] = 1; b.elements[3] = 2;

	int i,j;
	
	for (i=0;i<2;i++)
		for (j=0;j<2;j++)
			a.elements[i*a.width+j] = (float)(i+j);

	for (i=0;i<2;i++)
		for (j=0;j<2;j++)
			b.elements[i*b.width+j] = (float)(i+j);
	
	
	
	MatMul(a,b,c);

	/*for (int i=0;i<4;i++){
		printf("%f \n", a.elements[i]);
	}*/

	for (i=0;i<2;i++)
		for (j=0;j<2;j++)
		printf("%f \n", c.elements[i*c.width + j]);
			

	//for (i=0;i<2;i++){
	//	for (j=0;j<2;j++){
			

}
