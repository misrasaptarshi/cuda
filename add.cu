
#include <hip/hip_runtime.h>
#include<stdio.h>

//Device code

__global__ void addvec (float* a, float* b, float* c, int N)
{
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	if (i<N)
	 c[i] = a[i]+b[i];
}

//Host code

int main()
{
	int N = 10;
	size_t size = N*sizeof(float);

	//Allocate input vectors h_A and h_B in host memory
	float* h_a = (float*)malloc(size);
	float* h_b = (float*)malloc(size);
	float* h_c = (float*)malloc(size);

        //Initialize input vectors
	int i;
	
        for (i=0;i<N;i++){
		h_a[i] = i+1;
	}

	for (i=0;i<N;i++){
		h_b[i] = i+1;
	}

	//Allocate vectors in device memory
	float* d_a;
	hipMalloc(&d_a, size);
	float* d_b;
	hipMalloc(&d_b,size);
	float* d_c;
	hipMalloc(&d_c,size); 

	//Copy vectors from host memory to device memory
	hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

	//Invoke kernel
	int threads_per_block = 256;
	int blocks_per_grid = (N + threads_per_block - 1) / threads_per_block;
	addvec<<<blocks_per_grid, threads_per_block>>>(d_a, d_b, d_c, N);

	//Copy result from device memory to host memory
	//h_c contains the result in host memory
	hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

	for (i=0;i<N;i++){
		printf("%f \n",h_c[i]);
	}

	//Free device memory
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	//Free host memory
	hipFree(h_a);
	hipFree(h_b);
}

